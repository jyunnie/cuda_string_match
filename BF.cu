#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define DATASIZE 1000000
FILE *fp;
FILE *db;
hipError_t searchKeyword(unsigned int *result,unsigned  char **data,unsigned  char **keyword);
__global__ void searchKeywordKernel(unsigned int *result,unsigned char *data,unsigned  char *keyword)
{
 int i = threadIdx.x;
 // Detect the first matching character
 if (data[i] == keyword[0]) {
   // Loop through next keyword character
   for (int j=1; j<3; i++) {
     if (data[i+j] != keyword[j])
       break;
     else
     // Store the first matching character to the result list
       result[i] = 1;
   }
  }
}
int main()
{
 printf("Start\n");	
 unsigned char *line[DATASIZE];
 unsigned char *dbline[DATASIZE];
 char *datacp[DATASIZE];
 char *keywordcp[DATASIZE];
 //char data[DATASIZE];
 char *keyword[DATASIZE];
 unsigned int result[DATASIZE] = { 0 };
 // Set false value in result array
 memset(result, 0, DATASIZE);
 unsigned long i = 0;//
 unsigned long j= 0;
  
  fp=fopen("Taiwan_input.input","r");
  if(fp != NULL)
    {
      printf("reading input\n");
      while(fgets(line,sizeof(line),fp))
	{
	
	  strcpy(keyword[i],line);// seperated lines in st[]
	  i++;
	}
    }
  printf("complete!\n");
  fclose(fp);
  db=fopen("Taiwan_db.txt","r");

  if(db != NULL)
    {
      printf("reading db\n");
      while(fgets(dbline,sizeof(dbline),db))
	{
	  strcpy(datacp[j],dbline);// seperated dblines in st_db[]
	  j++;
	}
    }
  printf("complete!\n");
  fclose(db);
 // Generate input data
 
// Search keyword in parallel.
hipError_t cudaStatus = searchKeyword(result, datacp, keywordcp);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "addWithCuda failed!");
  return 1;
}
 
// Print out the string match result position
int total_matches = 0;
for (int i=0; i<DATASIZE; i++) {
  if (result[i] == 1) {
    printf("Character found at position % i\n", i);
    total_matches++;
  }
}
printf("Total matches = %d\n", total_matches);
// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceReset failed!");
  return 1;
}
system("pause");
return 0;
 
}
// Helper function for using CUDA to search a list of characters in parallel.
hipError_t searchKeyword(unsigned int *result,unsigned  char **data,unsigned char **keyword)
{
 unsigned char *dev_data = 0;
 unsigned char *dev_keyword = 0;
 unsigned int *dev_result = 0;
 hipError_t cudaStatus;
 // Choose which GPU to run on, change this on a multi-GPU system.
 cudaStatus = hipSetDevice(0);
 if (cudaStatus != hipSuccess) {
   fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
   goto Error;
 }
 // Allocate GPU buffers for result set.
 cudaStatus = hipMalloc((void**)&dev_result, DATASIZE * sizeof(unsigned int));
 if (cudaStatus != hipSuccess) {
   fprintf(stderr, "hipMalloc failed!");
   goto Error;
 }
 // Allocate GPU buffers for data set.
 cudaStatus = hipMalloc((void**)&dev_data, DATASIZE * sizeof(unsigned char));
 if (cudaStatus != hipSuccess) {
   fprintf(stderr, "hipMalloc failed!");
   goto Error;
 }
// Allocate GPU buffers for keyword.
cudaStatus = hipMalloc((void**)&dev_keyword, DATASIZE * sizeof(unsigned char));
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMalloc failed!");
  goto Error;
}
// Copy input data from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_data, data, DATASIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
// Copy keyword from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_keyword, keyword, DATASIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
// Launch a search keyword kernel on the GPU with one thread for each element.
searchKeywordKernel<<<1, DATASIZE>>>(dev_result, dev_data, dev_keyword);
// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
  goto Error;
}
// Copy result from GPU buffer to host memory.
cudaStatus = hipMemcpy(result, dev_result, DATASIZE * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
Error:
 hipFree(dev_result);
 hipFree(dev_data);
 hipFree(dev_keyword);
 
 return cudaStatus;
 }
