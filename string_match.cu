include<stdio.h>
#include<stdlib.h>
#define N 2048
#define Blocksize 16



int main()
{
  FILE *fp;
  int i,*input_file_index;
  char ch;
  
  fp=fopen("InputAnnotation.txt","r");
  while(!feof(fp))
    {
      ch = fgetc(fp);
      if(ch == '\n')
	{
	  i++;
	}
    }
  input_file_index = (int *)malloc(sizeof(i));
  
}

/*void fgetdata()
{
  //sprintf(FILENAME,"%s%.3d%s",__FILENAME, FILE_NUM, FILETYPE);
  
  if ((fp = fopen(FILENAME, "rb")) == NULL) {
    puts(FILENAME);
    puts("file open error!!");
  }
  
  fread(DATA_NUM, sizeof(int), 1, fp);
  printf("FILENAME: %s, DATA_NUM[0]: %d\n",FILENAME,DATA_NUM[0]);	
  fread(st, sizeof(int), DATA_NUM[0] * 6, fp); 
  for(int j=0; j<=DATA_NUM[0]*6;j++)
    {
      chr[j] = st[j*6];
      start[j] = st[j*6+1];
      end[j]= st[j*6+2];
      ref[j] = st[j*6+3];
      alt[j] = st[j*6+4];
    }	
  fclose(fp);
  
  }*/

void memory_init(int lines){
  int deviceCount;
  hipGetDevice(&deviceCount);
  hipMalloc((void**) &d_row);
  hipMalloc((void**) &d_T);
}

void calculation()
{
  int deviceCount;
  dim3 grid();
  dim3 threads();
  fgetdata();
  hipGetDevice(&deviceCount);
  hipMemset();
  hipMemcpy(hipMemcpyHostToDevice);
  string_match<<< grid,threads >>>(parameters);
 
  hipMemcpy(hipMemcpyDeviceToHost);
  sprintf(FILENAME,write);
  fwrite();
  fp=fopen();
  fclose(fp);
}



