include<stdio.h>
#include<stdlib.h>
#define N 2048
#define Blocksize 16



int main()
{
  FILE *fp;
  int i,*input_file_index;
  char ch;
  
  fp=fopen("InputAnnotation.txt","r");
  while(!feof(fp))
    {
      ch = fgetc(fp);
      if(ch == '\n')
	{
	  i++;
	}
    }
  input_file_index = (int *)malloc(sizeof(i));
  
}

/*void fgetdata()
{
  //sprintf(FILENAME,"%s%.3d%s",__FILENAME, FILE_NUM, FILETYPE);
  
  if ((fp = fopen(FILENAME, "rb")) == NULL) {
    puts(FILENAME);
    puts("file open error!!");
  }
  
  fread(DATA_NUM, sizeof(int), 1, fp);
  printf("FILENAME: %s, DATA_NUM[0]: %d\n",FILENAME,DATA_NUM[0]);	
  fread(st, sizeof(int), DATA_NUM[0] * 6, fp); 
  for(int j=0; j<=DATA_NUM[0]*6;j++)
    {
      chr[j] = st[j*6];
      start[j] = st[j*6+1];
      end[j]= st[j*6+2];
      ref[j] = st[j*6+3];
      alt[j] = st[j*6+4];
    }	
  fclose(fp);
  
  }*/

void memory_init(int lines){
  int deviceCount;
  hipGetDevice(&deviceCount);
  hipMalloc((void**) &d_row);
  hipMalloc((void**) &d_T);
}

void calculation()
{
  int deviceCount;
  dim3 grid();
  dim3 threads();
  fgetdata();
  hipGetDevice(&deviceCount);
  hipMemset();
  hipMemcpy(hipMemcpyHostToDevice);
  string_match<<< grid,threads >>>(parameters);
 
  hipMemcpy(hipMemcpyDeviceToHost);
  sprintf(FILENAME,write);
  fwrite();
  fp=fopen();
  fclose(fp);
}

}
// Launch a search keyword kernel on the GPU with one thread for each element.
searchKeywordKernel<<<1, DATASIZE>>>(dev_result, dev_data, dev_keyword);
// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
  goto Error;
}
// Copy result from GPU buffer to host memory.
cudaStatus = hipMemcpy(result, dev_result, DATASIZE * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
Error:
 hipFree(dev_result);
 hipFree(dev_data);
 hipFree(dev_keyword);
 
 return cudaStatus;
 
}
