#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<unistd.h>
#include<cuda_gl_interop.h>
#define N 2048
#define Blocksize 16
#define TOTAL_LINES 50000
//#include "kernel_calculation.cu"

//cuda memory allocation
char *cuda_lines;
char *cuda_wholedb;
unsigned int obj_size = TOTAL_LINES * sizeof(char);
unsigned int wholedb = 1000000 * sizeof(char);
char *cuda_output;
//unsigned int 

char output[N*N];
//FILES
FILE *fp;
FILE *db;
FILE *result;
char *st[4096],*st_db[10000000];
char line[511];
char dbline[511];

void cuda_init();
void calculation();
void cleanup();

int main()
{
  hipSetDevice(0);
  int i = 0;//
  int j= 0;

  fp=fopen("InputAnnotation.txt","r");
   if(fp != NULL)
    {
      char line[255];
      while(fgets(line,sizeof line,fp))
	{
	  st[i]=strdup(line);// seperated lines in st[]
	  i++;
	}
  fclose(fp);
  db=fopen("10mdbsnp.txt","r");
  if(db != NULL)
    {
      char line[255];
      while(fgets(dbline,sizeof dbline,db))
	{
	  st_db[j]=strdup(dbline);// seperated dblines in st_db[]
	  j++;
	}
  fclose(db)
  cuda_init();
  calculation();
  atexit(cleanup);
  hipDeviceReset();
  
  return 0;
}

void cuda_init(){
  int deviceCount;
  hipGetDevice(&deviceCount);
  hipMalloc((void**) &cuda_lines , obj_size);
  hipMalloc((void**) &cuda_wholedb,wholedb);
  hipMalloc((void**) &cuda_output,obj_size);
}

void calculation()
{
  int deviceCount;
  dim3 grid(4096);
  dim3 threads(256);
  
  hipGetDevice(&deviceCount);
  hipMemset(cuda_lines,0,obj_size);
  hipMemset(cuda_wholedb,0,wholedb);
  hipMemset(cuda_output,0,obj_size);
  hipMemcpy(cuda_lines,st,obj_size,hipMemcpyHostToDevice);
  hipMemcpy(cuda_wholedb,st_db,wholedb,hipMemcpyHostToDevice);

  string_match<<< grid,threads >>>(cuda_lines,cuda_wholedb,cuda_output);
 
  hipMemcpy(output,cuda_output,obj_size,hipMemcpyDeviceToHost);
  if((result = fopen("Output_Annotation.txt","wb")) == NULL){
      printf("FILE OPEN ERROR!! _WRITE");
      exit(1);
  }
  fwrite(output,sizeof(char),2048*2048),result);
 
  fclose(result);
}

void cleanup()
{
	printf("cleanup\n");
	//---------------------------------------------
	
	hipFree(cuda_lines);
	hipFree(cuda_wholedb);
	hipFree(cudaoutput);
}

__global__ void calculation(const char *substr,int len,int substrlen)
{

  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  const* char s1 = s; //text in device memory;
  const* char s2 = substr;//input text
  unsigned int yes = 1;
  int curr_marker = 0;

  if((len - shft)<substrlen)
  {
	res[shft]=0;
	return;
   }
	//scan the text in device memory,attempt to match pattern
   for(int i=shft;curr_marker <= substrlen && i<len;curr_marker++,i++){
	if(s2[curr_marker] && (s2[curr_marker]!=s1[i])){
		yes = 0;
		break;
	}
    }

   if(yes==1)
	res[shft] = yes;
}









