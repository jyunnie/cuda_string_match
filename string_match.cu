#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define DATASIZE 260
hipError_t searchKeyword(int *result, char *data, char *keyword);
__global__ void searchKeywordKernel(int *result, char *data, char *keyword)
{
 int i = threadIdx.x;
 // Detect the first matching character
 if (data[i] == keyword[0]) {
   // Loop through next keyword character
   for (int j=1; j<3; i++) {
     if (data[i+j] != keyword[j])
       break;
     else
     // Store the first matching character to the result list
       result[i] = 1;
   }
  }
}
int main()
{
 char data[DATASIZE];
 char keyword[2] = { 'K', 'L'};
 int result[DATASIZE] = { 0 };
 // Set false value in result array
 memset(result, 0, DATASIZE);
 // Generate input data
 int tmpindex = 65;
 for (int i=0; i<DATASIZE; i++) {
   data[i] = char(tmpindex);
   (tmpindex == 90 ? tmpindex = 65 : tmpindex++);
 }
 // Print the input character
for (int i=0; i<DATASIZE; i++)
  printf("i=%d,%c ", i, data[i]);
printf("\n");
// Search keyword in parallel.
hipError_t cudaStatus = searchKeyword(result, data, keyword);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "addWithCuda failed!");
  return 1;
}
 
// Print out the string match result position
int total_matches = 0;
for (int i=0; i<DATASIZE; i++) {
  if (result[i] == 1) {
    printf("Character found at position % i\n", i);
    total_matches++;
  }
}
printf("Total matches = %d\n", total_matches);
// cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceReset failed!");
  return 1;
}
system("pause");
return 0;
 
}
// Helper function for using CUDA to search a list of characters in parallel.
hipError_t searchKeyword(int *result, char *data, char *keyword)
{
 char *dev_data = 0;
 char *dev_keyword = 0;
 int *dev_result = 0;
 hipError_t cudaStatus;
 // Choose which GPU to run on, change this on a multi-GPU system.
 cudaStatus = hipSetDevice(0);
 if (cudaStatus != hipSuccess) {
   fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
   goto Error;
 }
 // Allocate GPU buffers for result set.
 cudaStatus = hipMalloc((void**)&dev_result, DATASIZE * sizeof(int));
 if (cudaStatus != hipSuccess) {
   fprintf(stderr, "hipMalloc failed!");
   goto Error;
 }
 // Allocate GPU buffers for result set.
 cudaStatus = hipMalloc((void**)&dev_data, DATASIZE * sizeof(char));
 if (cudaStatus != hipSuccess) {
   fprintf(stderr, "hipMalloc failed!");
   goto Error;
 }
// Allocate GPU buffers for keyword.
cudaStatus = hipMalloc((void**)&dev_keyword, DATASIZE * sizeof(char));
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMalloc failed!");
  goto Error;
}
// Copy input data from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_data, data, DATASIZE * sizeof(char), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
// Copy keyword from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_keyword, keyword, DATASIZE * sizeof(char), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
// Launch a search keyword kernel on the GPU with one thread for each element.
searchKeywordKernel<<<1, DATASIZE>>>(dev_result, dev_data, dev_keyword);
// cudaDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
  goto Error;
}
// Copy result from GPU buffer to host memory.
cudaStatus = hipMemcpy(result, dev_result, DATASIZE * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipMemcpy failed!");
  goto Error;
}
Error:
 hipFree(dev_result);
 hipFree(dev_data);
 hipFree(dev_keyword);
 
 return cudaStatus;
 
}
